
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>

int main() {

	int* vetor; //Declares a integer pointer
	int* vetor_d; //Declares a integer pointer
    int N = 10; //Declares and initializes N to 10

	int buffer_size = sizeof(int)*N; //Number of bytes in our array
	vetor = (int*) malloc (buffer_size); //Allocates the host vector
	hipMalloc((void**) &vetor_d, buffer_size); //Allocates the device vector
	
	for (int i=0; i<N; i++) {
		vetor[i] = N - i;
	}

    //Copies from the host to the device
	hipMemcpy( vetor_d, vetor, buffer_size, hipMemcpyHostToDevice ); 
    //Sets the values of the device vector to zero
	hipMemset( vetor_d, 0, buffer_size); 
    //Copies the vector back from the device to the host
	hipMemcpy( vetor, vetor_d, buffer_size, hipMemcpyDeviceToHost ); 
	
	for (int i=0; i<N; i++) {
		std::cout << "vetor em " << i << ": " << vetor[i] << std::endl;
	}
	
	return 0;
}
	
